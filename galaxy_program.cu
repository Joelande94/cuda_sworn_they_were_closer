/*
    CUDA good to knows:
        Basics:
            Per thread:
                registers (fast)
                local memory (off-chip [still on the GPU though], slow)

            Per block:
                multiple threads
                shared memory (semi-fast)

            Per GPU:
                Multiple kernels that each run multiple blocks
                Global memory (off-chip [still on the GPU though], slow)


            Threads are executed by thread processors
            
            Threads reside in thread blocks
            
            Thread blocks are executed by multiprocessors

            Several concurrent thread blocks can reside on one multiprocessor
                Limited by multiprocessor resources (shared memory and registers)

            A kernel is launched as a grid of thread blocks. Only one kernel can execute on a device at a time.
        Advanced:
            cudaMemcpy(dst, src, size, direction)
                blocks CPU thread.
            

    Compiler tips:
        nvcc <filename>.cu [-o <executable>]
            Builds release mode

        nvcc -g <filename>.cu
            Builds debug mode
            Can debug host code but not device code

        nvcc -deviceemu <filename>.cu
            Builds device emulation mode
            All code runs on CPU, no debug symbols

        nvcc -deviceemu -g <filename>.cu
            Builds debug device emulation mode
            All code runs on CPU, with debug symbols

    Tips and tricks:
        If our arrays A,B,C are shorter than 1024 elements, N < 1024, then
            – one thread block is enough
            – N threads in the thread block
        If our arrays are longer than 1024, then
            – Choose the number of threads in the thread blocks to be
            integer*32
            – Calculate how many thread blocks you need
            – There will be some threads that should do nothing
        Why multiples of 32?
            – Threads are executed synchronously in bunches of 32 =
            warp
            – All threads must have their data ready before the warp runs
            – Cache lines are 4 B x warp size = 128 B
            – GPU resources can be fully utilized when these parameters
            are used
        # of blocks = ceil(N/threadsInBlock)
                    = (N+threadsInBlock-1)/threadsInBlock



Compile:
nvcc -o galaxy galaxy_program.cu -res-usage

Run:
time ./galaxy

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

using namespace std;


// Declare functions and classes that are below main.
class GalaxyFile{
public:
    int number_of_galaxies;
    float *alphas, *deltas;

    GalaxyFile(){}

    GalaxyFile(int num, float *as, float *ds)
    {
        number_of_galaxies = num;
        alphas = as;
        deltas = ds;
    }
};
void print_omegas(float*, int);
void write_omegas_to_file(string, float*);
void write_histogram_to_file(string, int*);
void print_histogram(string, int*, int);
GalaxyFile readFile(string);


// Define some useful macros
#define BIN_WIDTH 0.25f
#define BIN_MIN 0.0f
#define BIN_MAX 180.0f
#define NUMBER_OF_BINS (int)(BIN_MAX*(1.0f/BIN_WIDTH))

// Google is your friend.
#define ARCMINS_TO_RADIANS 0.000290888209f
#define RADIANS_TO_DEGREES 57.295779513f


__global__
void angle_between_galaxies(float *alphas1, float *deltas1, float *alphas2, float *deltas2, int *gpu_hist){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    
    if(idx < 100000){
        for(int i=0; i<100000; i++){
                float angle = 0.0f;
                // Don't do duplicates
                if( alphas1[i] != alphas2[idx] && deltas1[i] != deltas2[idx] ) {
                    float x = sin(deltas1[i]) * sin(deltas2[idx]) + cos(deltas1[i]) * cos(deltas2[idx]) * cos(alphas1[i] - alphas2[idx]);
                    angle = acosf(fmaxf(-1.0f, fminf(x, 1.0f))) * RADIANS_TO_DEGREES;
                }

                int ix = (int)(floor(angle * (1.0f/BIN_WIDTH))) % NUMBER_OF_BINS;
                __syncthreads();
                atomicAdd(&gpu_hist[ix], 1);
        }
    }
}

int* calculate_histogram(GalaxyFile galaxies1, GalaxyFile galaxies2){
    // Declare and allocate memory for histogram arrays that will be accessible on CPU
    float galaxy_array_size = galaxies1.number_of_galaxies * sizeof(float);
    float histogram_size = NUMBER_OF_BINS * sizeof(int);

    int *histogram;
    int *total_histogram;
    histogram = (int *) malloc(NUMBER_OF_BINS*sizeof(int));
    total_histogram = (int *) malloc(NUMBER_OF_BINS*sizeof(int));

    memset(total_histogram, 0, NUMBER_OF_BINS*sizeof(int));

    // Declare angle arrays that will be accessible on GPU
    float *gpu_alphas1;
    float *gpu_deltas1;
    float *gpu_alphas2;
    float *gpu_deltas2;
    int *gpu_histogram;

    // Allocate memory on GPU for angle arrays
    hipMalloc((void**) &gpu_alphas1, galaxy_array_size);
    hipMalloc((void**) &gpu_deltas1, galaxy_array_size);
    hipMalloc((void**) &gpu_alphas2, galaxy_array_size);
    hipMalloc((void**) &gpu_deltas2, galaxy_array_size);
    hipMalloc((void**) &gpu_histogram, NUMBER_OF_BINS*sizeof(int));

	// Copy angles from CPU onto GPU
	hipMemcpy(gpu_alphas1, galaxies1.alphas, galaxy_array_size, hipMemcpyHostToDevice);
	hipMemcpy(gpu_deltas1, galaxies1.deltas, galaxy_array_size, hipMemcpyHostToDevice);
	hipMemcpy(gpu_alphas2, galaxies2.alphas, galaxy_array_size, hipMemcpyHostToDevice);
	hipMemcpy(gpu_deltas2, galaxies2.deltas, galaxy_array_size, hipMemcpyHostToDevice);
	hipMemcpy(gpu_histogram, histogram, galaxy_array_size, hipMemcpyHostToDevice);
    
    int warp_size = 32;
    int threadsInBlock = 11 * warp_size;
    int blocksInGrid = ceil((galaxies1.number_of_galaxies + threadsInBlock) / threadsInBlock);

	// Define the grid size (blocks per grid)
    dim3 dimGrid(blocksInGrid);
    
	// Define block size (threads per block)
	dim3 dimBlock(threadsInBlock);

    // Write histogram full of zeros
    hipMemset(gpu_histogram, 0, histogram_size);

    // Calculate angles between galaxies1[i] and every galaxy in galaxies2
    angle_between_galaxies<<<dimGrid, dimBlock>>>(gpu_alphas1, gpu_deltas1, gpu_alphas2, gpu_deltas2, gpu_histogram);
    
    // Copy result histogram into CPU histogram
    hipMemcpy(histogram, gpu_histogram, histogram_size, hipMemcpyDeviceToHost);
    
	// Free all the memory we allocated on the GPU
	hipFree( gpu_alphas1 );
	hipFree( gpu_deltas1 );
	hipFree( gpu_alphas2 );
	hipFree( gpu_deltas2 );
    hipFree( gpu_histogram );

    return histogram;
}

float* calculate_omegas(int* DD, int* DR, int* RR){
    float* omegas;
    omegas = (float *) malloc(NUMBER_OF_BINS*sizeof(float));

    for(int i=0; i<NUMBER_OF_BINS; i++){
        if(RR[i] != 0.0f){
            omegas[i] = (DD[i] - 2.0f*DR[i] + RR[i]) / RR[i];
        }else{
            omegas[i] = 0.0f;
        }
    }
    return omegas;
}

// CUDA program that calculates distribution of galaxies
int main()
{
    // Read files and store data in GalaxyFile classes.
    GalaxyFile galaxies1;
    GalaxyFile galaxies2;
    galaxies1 = readFile("test_data/flat_100k_arcmin.txt");
    galaxies2 = readFile("test_data/data_100k_arcmin.txt");

    int* DD_hist = calculate_histogram(galaxies1, galaxies1);
    int* DR_hist = calculate_histogram(galaxies1, galaxies2);
    int* RR_hist = calculate_histogram(galaxies2, galaxies2);

    print_histogram("DD", DD_hist, 20);
    print_histogram("DR", DR_hist, 20);
    print_histogram("RR", RR_hist, 20);

    write_histogram_to_file("dd_histogram.txt", DD_hist);
    write_histogram_to_file("dr_histogram.txt", DR_hist);
    write_histogram_to_file("rr_histogram.txt", RR_hist);

    float* omegas = calculate_omegas(DD_hist, DR_hist, RR_hist);
    print_omegas(omegas, 15);
    write_omegas_to_file("omegas.txt", omegas);

	return EXIT_SUCCESS;
}


/* UTILITY FUNCTIONS/CLASSES BELOW */
GalaxyFile readFile(string filename)
{
    ifstream infile(filename);
    int number_of_galaxies;

    // Read first line which is the number of galaxies that's stored in the file.
    infile >> number_of_galaxies;

    float galaxy_array_size = number_of_galaxies * sizeof(float);

    float *alphas, *deltas;
    alphas = (float*) malloc(galaxy_array_size);
    deltas = (float*) malloc(galaxy_array_size);

    float alpha;
    float delta;

    // Read arc minute angles for each galaxy
    // Then convert those angles to radians and store those in alphas and deltas arrays
    for(int i=0; i<number_of_galaxies; i++) {
        infile >> alpha >> delta;

        alphas[i] = alpha * ARCMINS_TO_RADIANS;
        deltas[i] = delta * ARCMINS_TO_RADIANS;
    }
    infile.close();

    GalaxyFile galaxyFile(number_of_galaxies, alphas, deltas);
    return galaxyFile;
}

void print_omegas(float* omegas, int bins_to_print){
    for (int i=0; i<NUMBER_OF_BINS; i++){
        if(omegas[i] != 0.0f && i < bins_to_print){
            printf("omegas[%d]: %f\n", i, omegas[i]);
        }
    }
}

void print_histogram(string label, int *histogram, int bins_to_print){
    long long galaxies_counted = 0;
    // Print each bucket bin that has 1 or more galaxy-pair-angle in it.
    for (int i=0; i<NUMBER_OF_BINS; i++) {
        float bucket_min = (float)i / (1.0f/BIN_WIDTH);
        float bucket_max = (float)i / (1.0f/BIN_WIDTH) + BIN_WIDTH;
        int bucket_value = histogram[i];

        galaxies_counted += histogram[i];
        if(bucket_value > 0 && i < bins_to_print){
            printf("[%f, %f]: %d\n", bucket_min, bucket_max, bucket_value);
        }
    }

    cout << "Galaxy pairs counted in " << label << ": " << galaxies_counted << endl;
}

void write_omegas_to_file(string filename, float* omegas){
    ofstream file;
    file.open("output/"+filename);

    for (int i=0; i<NUMBER_OF_BINS; i++){
        file << omegas[i];
        if(i<NUMBER_OF_BINS-1) file << "\n";
    }
    file.close();
}

void write_histogram_to_file(string filename, int* histogram){
    ofstream file;
    file.open("output/"+filename);
    
    for (int i=0; i<NUMBER_OF_BINS; i++){
        file << histogram[i]; 
        if(i<NUMBER_OF_BINS-1) file << "\n";
    }
    file.close();
}